
#include <hip/hip_runtime.h>
extern "C" __global__ void kRejectMinimizationStep_kernel( int numAtoms, float4 *posq, float4 *oldPosq ) {
	for( int atom = threadIdx.x + blockIdx.x * blockDim.x; atom < numAtoms; atom += blockDim.x * gridDim.x ) {
		posq[atom] = oldPosq[atom];
	}
}

extern "C" __global__ void kAcceptMinimizationStep_kernel( int numAtoms, float4 *posq, float4 *oldPosq ) {
	for( int atom = threadIdx.x + blockIdx.x * blockDim.x; atom < numAtoms; atom += blockDim.x * gridDim.x ) {
		oldPosq[atom] = posq[atom];
	}
}
